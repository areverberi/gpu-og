#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "string.h"
#include <stddef.h>
#include <vector>
#define _USE_MATH_DEFINES
#include "math.h"
#include "hip/hip_runtime_api.h"
#include<hip/hip_runtime.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/tuple.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>
#include <thrust/gather.h>

#define NUM_PARTICLES 1000
#define PART_PER_THREAD 100
#define ALPHA1 0.05f
#define ALPHA2 0.057f
#define ALPHA3 0.087f
#define ALPHA4 0.05f

typedef thrust::device_vector<float>::iterator floatIterator;
typedef thrust::tuple<floatIterator, floatIterator, floatIterator> floatIterTuple;
typedef thrust::zip_iterator<floatIterTuple> zipIteratorFloatTuple;

struct pseudorgnorm
{
	float a,b;
	__host__ __device__ pseudorgnorm(float _a=0.0f, float _b=1.0f): a(_a), b(_b) {};
	__host__ __device__ float operator()(const unsigned int n)const
	{
		thrust::default_random_engine rng;
		thrust::normal_distribution<float>dist(a,b);
		rng.discard(n);
		return dist(rng);
	}
};

struct pseudorg
{
	float a,b;
	__host__ __device__ pseudorg(float _a=0.0f, float _b=1.0f): a(_a), b(_b) {};
	__host__ __device__ float operator()(const unsigned int n)const
	{
		thrust::default_random_engine rng;
		thrust::uniform_real_distribution<float>dist(a,b);
		rng.discard(n);
		return dist(rng);
	}
};

template <typename T> 
struct lin_to_row_index : public thrust::unary_function<T, T>
{
	T C;
	__host__ __device__ lin_to_row_index(T _C): C(_C) {}
	__host__ __device__ T operator()(T i)
	{
		return i/C;
	}
};

template <typename T> 
struct cos_v : public thrust::unary_function<T, T>
{
	__host__ __device__ T operator()(T i)
	{
		return __cosf(i);
	}
};

template <typename T> 
struct sin_v : public thrust::unary_function<T, T>
{
	__host__ __device__ T operator()(T i)
	{
		return __sinf(i);
	}
};

//texture <float, 2, hipReadModeElementType> radius;
//texture <float, 2, hipReadModeElementType> angle;
//texture <float, 2, hipReadModeElementType> sensor_model;

//__constant__ float x;
//__constant__ float y;
//__constant__ float theta;
__constant__ int mapW;
__constant__ int mapH;
__constant__ float resolution;
__constant__ float range_max;

__device__ float fatomicMin(float *addr, float value)
{
	float old = *addr, assumed;
	if(old <= value) return old;
	do
	{
		assumed = old;
		old = atomicCAS((unsigned int*)addr, __float_as_int(assumed), __float_as_int(value));
	}while(old!=assumed);
	return old;
}
bool readLine(std::ifstream& file, std::vector<int>& numScans, std::vector<std::vector<float>>& scans, std::vector<float>& x, std::vector<float>& y, std::vector<float>& theta)
{
	std::string line_type;
	file>>line_type;
	if(line_type=="#" || line_type=="PARAM" || line_type=="ODOM" || line_type=="NEFF")
	{
		std::string skip;
		std::getline(file, skip);
		return true;
	}
	if(line_type=="FLASER")
	{
		int num;
		file>>num;
		numScans.push_back(num);
		std::vector<float> scan(num);
		for(unsigned int i=0; i<num; ++i)
		{
			float s;
			file>>s;
			scan.push_back(s);
		}
		scans.push_back(scan);
		float t;
		file>>t;
		x.push_back(t);
		file>>t;
		y.push_back(t);
		file>>t;
		theta.push_back(t);
		std::string rem;
		std::getline(file, rem);
		return true;
	}
	return false;
}
bool loadLog(std::string filename, std::vector<int>& numScans, std::vector<std::vector<float>>& scans, std::vector<float>& x, std::vector<float>& y, std::vector<float>& theta)
{
	std::ifstream file(filename.c_str());
	if(!file.is_open())
		return false;
	while(readLine(file, numScans, scans, x, y, theta));
	return true;
}
char* mystrsep(char** stringp, const char* delim)
{
  char* start = *stringp;
  char* p;

  p = (start != NULL) ? strpbrk(start, delim) : NULL;

  if (p == NULL)
  {
    *stringp = NULL;
  }
  else
  {
    *p = '\0';
    *stringp = p + 1;
  }

  return start;
}
__global__ void __launch_bounds__(1024) initMap(float* map, int w, int h, size_t pitch, int w_th, int h_th){
	unsigned int idx=blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int idy=blockIdx.y*blockDim.y+threadIdx.y;
	//unsigned int w_th=w/(blockDim.x*gridDim.x);
	//unsigned int h_th=h/(blockDim.y*gridDim.y);
	for(unsigned int i=0; i<h_th; ++i)
	{
		for(unsigned int j=0; j<w_th; ++j)
		{
			unsigned int x=idx*w_th+j;
			unsigned int y=idy*h_th+i;
			if(x<w && y<h)
			{
				map[x+y*pitch]=-1.0f;
			}
		}
	}
    __syncthreads();
}

__device__ void getCoordsBresenham(float *coords, float * range, float * x_o, float * y_o, float * theta_o, int coord_sys=0)
{
	__shared__ int x1, y1, x2, y2;
	__shared__ float delta_x, delta_y, m;
	__shared__ int sign_delta_x, sign_delta_y;
	__shared__ float theta_b;

	if(threadIdx.x==0)
	{
		theta_b=*theta_o+blockIdx.x*M_PI/359-M_PI_2;
		float s;
		float c;
		__sincosf(theta_b, &s, &c);
		//mapW/H is offset, 0.1f is resolution
		x1=(int)floorf(mapW/2+*x_o/resolution);
		y1=(int)floorf(mapH/2+*y_o/resolution);
		//0.1f for wall thickness, if needed, add to range before mul
		x2=(int)floorf(mapW/2+(*x_o +(*range+0.1f)*c)/resolution);
		y2=(int)floorf(mapH/2+(*y_o +(*range+0.1f)*s)/resolution);
		delta_x=(float)(x2-x1);
		delta_y=(float)(y2-y1);
		/*
		sign_delta_x=1;
		if(delta_x<0)sign_delta_x=-1;
		sign_delta_y=1;
		if(delta_y<0)sign_delta_y=-1;
		*/
		sign_delta_x=copysignf(1, delta_x);
		sign_delta_y=copysignf(1, delta_y);
	}
	__syncthreads();
	if(*range<range_max)
	{
		int current_x, current_y, pos;
		if(coord_sys==0)
			pos=threadIdx.x;
		else
			pos=blockIdx.y;
		if(fabs(delta_y)>fabs(delta_x))
		{
			m=delta_x/delta_y;
			current_y=y1+sign_delta_y*pos;
			current_x=x1+rintf(m*(current_y-y1));
            //current_x=x1+floorf(0.4999999f+m*(current_y-y1));
		}
		else
		{
			m=delta_y/delta_x;
			current_x=x1+sign_delta_x*pos;
			current_y=y1+rintf(m*(current_x-x1));
            //current_y=y1+floorf(0.4999999f+m*(current_x-x1));
		}
		coords[0]=current_x;
		coords[1]=current_y;
	}
	else
	{
		coords[0]=-1;
		coords[1]=-1;
	}
	if(coords[0]>=0 && coords[0]<mapW && coords[1]>=0 && coords[1]<mapH)
	{
		coords[2]=hypotf(coords[0]-x1, coords[1]-y1)*resolution;
	}
	else
	{
		coords[2]=-1;
	}
}

__global__ void computeMatchScores(float * x_part, float * y_part, float * theta_part, float * scan_gpu, float *map, size_t pitch, float * scores)
{
	//__shared__ float range;
	////__shared__ float x;
	////__shared__ float y;
	////__shared__ float theta;
	//__shared__ float true_range;
	////__shared__ float computed_ranges[256];
	//float coords[3];
	//if(threadIdx.x==0)
	//{
	//	range=range_max-0.0001f;
	//	//true_range=scan_gpu[blockIdx.x];
	//}
	//__syncthreads();
	//for(int i=threadIdx.x*PART_PER_THREAD; i<threadIdx.x*PART_PER_THREAD+PART_PER_THREAD; ++i)
	//{
	//	float x=x_part[i];
	//	float y=y_part[i];
	//	float theta=theta_part[i];
	//	getCoordsBresenham(coords, &range, &x, &y, &theta, 1);
	//	if(coords[2]>=0.0f && map[(int)coords[0]+(int)coords[1]*pitch]>0.5f)
	//	//computed_ranges[threadIdx.x]=coords[2];
	//		fatomicMin(&scores[i], coords[2]);
	//}
	//__syncthreads();
	//computed_ranges[threadIdx.x]=range_max;	
	/*int threadsInB=blockDim.x;
	while(threadsInB>1)
	{
		int halfTh=(threadsInB >> 1);
		if(threadIdx.x<halfTh)
		{
			int thread2=threadIdx.x+halfTh;
			float temp=computed_ranges[thread2];
			if(temp<computed_ranges[threadIdx.x])
				computed_ranges[threadIdx.x]=temp;
		}
		__syncthreads();
		threadsInB=halfTh;
	}
	__syncthreads();*/
	//if(threadIdx.x==0)
	//{
	//	//float score=(true_range-computed_ranges[0])*(true_range-computed_ranges[0])/(true_range*computed_ranges[0]);
	//	float score=(true_range-computed_range)*(true_range-computed_range)/(true_range*computed_range);
	//	scores[blockIdx.x+blockIdx.y*gridDim.y]=score;
	//}
	__shared__ float x,y,theta;
	__shared__ unsigned int score;
	if(threadIdx.x==0)
	{
		x=x_part[blockIdx.x];
		y=y_part[blockIdx.x];
		theta=theta_part[blockIdx.x];
	}
	__syncthreads();
	float range=scan_gpu[threadIdx.x];
	float theta_t=theta+threadIdx.x*M_PI/359-M_PI_2;
	float s, c;
	__sincosf(theta_t, &s, &c);
	int x_hit=(int)floorf(mapW/2+(x+(range+0.1f)*c)/resolution);
	int y_hit=(int)floorf(mapH/2+(y+(range+0.1f)*s)/resolution);
	if(x_hit>=0 && x_hit<mapW && y_hit>=0 && y_hit<mapH)
		if(map[x_hit+y_hit*pitch]>0.5f)
			atomicInc(&score, 0);
	__syncthreads();
	if(threadIdx.x==0)
	{
		scores[blockIdx.x]=(float)score;
	}
}

__global__ void updateMapBresenham(float *map, size_t pitch, float *scan_gpu, float x, float y, float theta){
	__shared__ float range;
	float coords[3];
	if(threadIdx.x==0)
	{
		range=scan_gpu[blockIdx.x];
	}
	getCoordsBresenham(coords, &range, &x, &y, &theta);
	//printf("coords:%d %d\n", coords[0], coords[1]);
	if(coords[2]>=0)
	{
		//0.1f because going from grid (10cm cell) to meters
		float d=coords[2];
		int current_x=(int)coords[0];
		int current_y=(int)coords[1];
		//divide by 100 because rmax is #of cells, ie 500->turn to meters
		//float k=1-(d/rmax)*(d/rmax)/100;
		//float k=1;
        //float k=0.6;
		//float s=0.00001425*range*range;
		//float s=0.4;
        //float s=0.6;
		//float expon=((d-range)/s)*((d-range)/s);
		float prob;

		if(d<range)
		{
			//sensor model
			//prob=0.3+(k/s*__frsqrt_rn(s)+0.2)*__expf(-0.5*expon);
			if(d<1.0f)
			prob=0.45f;
			else
			prob=0.45f+(d-1.0f)/6.4f*(0.5f-0.45f);
		}
		else
		{
			//sensor model
			//prob=0.5+k/s*__frsqrt_rn(s)*__expf(-0.5*expon);
					
			if(d<1.0f)
			prob=0.75f;
			else
			prob=0.75f+(d-1.0f)/6.4f*(0.5f-0.75f);
					
		}
		//map[current_x+current_y*pitch]+=__logf(prob/(1-prob));
				
		if (d<=range+0.1f && d<=6.4f)
		{
			float pr=map[current_x+current_y*pitch];
			if(pr==-1.0f)
				pr=0.5f;
			map[current_x+current_y*pitch]=1.0f-1.0f/(1.0f+prob/(1.0f-prob)*pr/(1.0f-pr));

			//printf("-------------------------------------------------------------------updating map\n");  
		}
	}
	//if(threadIdx.x==0)
	//{
	//	range=scan_gpu[blockIdx.x];
	//	theta_b=theta+blockIdx.x*M_PI/359-M_PI_2;
	//	float s;
	//	float c;
	//	__sincosf(theta_b, &s, &c);
	//	//mapW/H is offset, 0.1f is resolution
	//	x1=(int)floorf(mapW/2+x/resolution);
	//	y1=(int)floorf(mapH/2+y/resolution);
	//	//0.1f for wall thickness, if needed, add to range before mul
	//	x2=(int)floorf(mapW/2+(x+(range+0.1f)*c)/resolution);
	//	y2=(int)floorf(mapH/2+(y+(range+0.1f)*s)/resolution);
	//	delta_x=(float)(x2-x1);
	//	delta_y=(float)(y2-y1);
	//	/*
	//	sign_delta_x=1;
	//	if(delta_x<0)sign_delta_x=-1;
	//	sign_delta_y=1;
	//	if(delta_y<0)sign_delta_y=-1;
	//	*/
	//	sign_delta_x=copysignf(1, delta_x);
	//	sign_delta_y=copysignf(1, delta_y);
	//}
	//__syncthreads();
	//if(range<range_max)
	//{
	//	int current_x, current_y;
	//	if(fabs(delta_y)>fabs(delta_x))
	//	{
	//		m=delta_x/delta_y;
	//		current_y=y1+sign_delta_y*threadIdx.x;
	//		current_x=x1+rintf(m*(current_y-y1));
 //           //current_x=x1+floorf(0.4999999f+m*(current_y-y1));
	//	}
	//	else
	//	{
	//		m=delta_y/delta_x;
	//		current_x=x1+sign_delta_x*threadIdx.x;
	//		current_y=y1+rintf(m*(current_x-x1));
 //           //current_y=y1+floorf(0.4999999f+m*(current_x-x1));
	//	}
	//	if(current_x>=0 && current_x<mapW && current_y>=0 && current_y<mapH)
	//	{
	//		//0.1f because going from grid (10cm cell) to meters
	//		float d=hypotf(current_x-x1, current_y-y1)*resolution;
	//		//divide by 100 because rmax is #of cells, ie 500->turn to meters
	//		//float k=1-(d/rmax)*(d/rmax)/100;
	//		//float k=1;
 //           //float k=0.6;
	//		//float s=0.00001425*range*range;
	//		//float s=0.4;
 //           //float s=0.6;
	//		//float expon=((d-range)/s)*((d-range)/s);
	//		float prob;

	//		if(d<range)
	//		{
	//			//sensor model
	//			//prob=0.3+(k/s*__frsqrt_rn(s)+0.2)*__expf(-0.5*expon);
	//			if(d<1.0f)
	//			prob=0.45f;
	//			else
	//			prob=0.45f+(d-1.0f)/6.4f*(0.5f-0.45f);
	//		}
	//		else
	//		{
	//			//sensor model
	//			//prob=0.5+k/s*__frsqrt_rn(s)*__expf(-0.5*expon);
	//				
	//			if(d<1.0f)
	//			prob=0.75f;
	//			else
	//			prob=0.75f+(d-1.0f)/6.4f*(0.5f-0.75f);
	//				
	//		}
	//		//map[current_x+current_y*pitch]+=__logf(prob/(1-prob));
	//			
	//		if (d<=range+0.1f && d<=6.4f)
	//		{
	//			float pr=map[current_x+current_y*pitch];
	//			if(pr==-1.0f)
	//				pr=0.5f;
	//			map[current_x+current_y*pitch]=1.0f-1.0f/(1.0f+prob/(1.0f-prob)*pr/(1.0f-pr));

	//			//printf("-------------------------------------------------------------------updating map\n");  
	//		}
	//	}
	//	else
	//	{
	//		//printf("%d %d\n", current_x, current_y); 
	//	}
	//}
	//else
	//{
	//	//printf("range: %d \n", range);
	//}
}
//__global__ void __launch_bounds__(1024) updateMap(float x, float y, float theta, float* map, float* scan_gpu, size_t pitch, int mapW, int mapH, float rmax){
//    __shared__ float scan[360];
//	/*first 360 threads load scan*/
//	unsigned int scanperthread=360/(blockDim.x*blockDim.y);
//	if(scanperthread>1){
//		unsigned int ind=(threadIdx.x*blockDim.x+threadIdx.y)*scanperthread;
//		unsigned int off;
//		for(off=0; off<scanperthread; off++){
//			if(ind+off<360)
//				scan[ind+off]=scan_gpu[ind+off];
//		}
//	}
//	else{
//		unsigned int ind=threadIdx.x*blockDim.x+threadIdx.y;
//		if(ind<360){
//			scan[ind]=scan_gpu[ind];
//		}
//	}
//	//printf("scan loaded\n");
//	__syncthreads();
//	float x_local_lu=(blockIdx.x*blockDim.x+threadIdx.x)*1.0/(gridDim.x*blockDim.x)*rmax;
//    float y_local_lu=(blockIdx.y*blockDim.y+threadIdx.y)*1.0/(gridDim.y*blockDim.y)*rmax;
//	/*to fix: the 10.0 should be s_m_resolution*/
//	//float val=tex2D(sensor_model, tex2D(radius, x_local_lu, y_local_lu)*10.0, scan[(int)rint(tex2D(angle, x_local_lu, y_local_lu))]*10.0);
//	float val=tex2D(sensor_model, scan[(int)rint(tex2D(angle, x_local_lu, y_local_lu))]*10.0, tex2D(radius, x_local_lu, y_local_lu)*10.0);
//	//if(tex2D(radius, x_local_lu, y_local_lu)>scan[(int)rint(tex2D(angle, x_local_lu, y_local_lu))])
//	//	printf("val:%f\n", val);
//	//printf("angle:%d\n", (int)rint(tex2D(angle, x_local_lu, y_local_lu)));
//	//printf("val:%f\n", val);
//	if (val!=0.5f)
//	{
//		float x_local=x_local_lu-rmax/2;
//		float y_local=rmax/2-y_local_lu;
//		x_local=x_local*__cosf(theta)-y_local*__sinf(theta);
//		y_local=x_local*__sinf(theta)+y_local*__cosf(theta);
//		//int x_map=(int)rint(x_local*cosf(theta)+y_local*sinf(theta)-x+mapW/2.0);
//		//int y_map=(int)rint(-x_local*sinf(theta)+y_local*cosf(theta)-y+mapH/2.0);
//		int x_map_cell=(int)rint(x_local+x*10.0f+mapW/2.0);
//		int y_map_cell=(int)rint(-(y_local+y*10.0f-mapH/2.0));
//		/*if(x_map_cell<0 || y_map_cell<0)
//		printf("%f %f\n", x_map_cell, y_map_cell);
//		*/if(x_map_cell<mapH && y_map_cell<mapW ){
//			//no size difference between local and global cells, otherwise you'd need to divide by global cell size to get map cell
//			//int x_map_cell=(int)rint(x_map);
//			//int y_map_cell=(int)rint(y_map);
//			if(scan[(int)rint(tex2D(angle, x_local_lu, y_local_lu))]>0.0)
//			{
//				size_t index=x_map_cell*pitch+y_map_cell;
//				//map[index]=0.5f*val+0.5f*map[index];
//				map[index]=1-1/(1+map[index]/(1-map[index])*val/(1-val));
//			}
//			/*
//			if(scan[(int)rint(tex2D(angle, x_local_lu, y_local_lu))]>0.0){}
//			if(map[index]<0.0f)
//			map[index]=val;
//			else
//			map[index]=0.5f*val+0.5f*map[index];
//			*/
//		} 
//	}
//    __syncthreads();
//}

int main(int argc, char** argv){
	/*float *r;
	float *a;
	float *s_m;*/
	/*size of the matrix in cells*/
	int local_size=500;
	//int map_size=1000;
    int map_size=1600;
	float cell_dim=0.1;
	//int map_size_x=1600;
	//int map_size_y=880;
	//float s_m_resolution=10.0;
	//r=(float*)malloc(sizeof(float)*local_size*local_size);
	//a=(float*)malloc(sizeof(float)*local_size*local_size);
	//s_m=(float*)malloc(sizeof(float)*local_size*local_size*(int)(s_m_resolution*s_m_resolution));
	//int loopX=0;
	//int loopY=0;
	///*initialization of lookups for radius, angle and sensor model*/
	//for(loopY=0; loopY<local_size*s_m_resolution; loopY++){
	//	for(loopX=0; loopX<local_size*s_m_resolution; loopX++){
	//		if(loopX<local_size && loopY<local_size){
	//			float x_cell=loopX*cell_dim+cell_dim/2.0f-local_size*cell_dim/2.0f;
	//			float y_cell=-loopY*cell_dim+cell_dim/2.0f+local_size*cell_dim/2.0f;
	//			r[loopY*local_size+loopX]=hypotf(x_cell, y_cell);
	//			a[loopY*local_size+loopX]=(atan2(y_cell, x_cell)+M_PI)/M_PI*180.0f;
	//			//a[loopY*local_size+loopX]=atan2(y_cell, x_cell)/M_PI*180.0f;
	//		}
	//		if (abs(loopX-loopY)<s_m_resolution/2.0){
	//			s_m[loopY*local_size*((int)s_m_resolution)+loopX]=0.95f;
	//		}
	//		else{
	//			if (loopY<loopX){
	//				s_m[loopY*local_size*((int)s_m_resolution)+loopX]=0.05f;
	//			}
	//			else{ 
	//				/*
	//				float min=(local_size*10<loopX+s_m_resolution?local_size*10:loopX+s_m_resolution);
	//				if (loopY> min){
	//					s_m[loopX*local_size*10+loopY]=0.5f;
	//				}
	//				*/
	//				s_m[loopY*local_size*((int)s_m_resolution)+loopX]=0.5f;
	//			}
	//		}
	//	}
	//}
	///*setting filter mode for the textures. It's linear for radius and angle so I get interpolation "for free"*/
	////printf("radius 0:%f\n", r[0]);
	////printf("angle 0:%f\n", a[0]);
	////getchar();
	//sensor_model.filterMode=hipFilterModePoint;
	//radius.filterMode=hipFilterModeLinear;
	//angle.filterMode=hipFilterModeLinear;	

	/*creating the cudaArrays that will contain the textures*/
	hipChannelFormatDesc cf=hipCreateChannelDesc<float>();
	/*hipArray *r_gpu;
	checkCudaErrors(hipMallocArray(&r_gpu, &cf, local_size, local_size));
	checkCudaErrors(hipMemcpyToArray(r_gpu, 0, 0, r, sizeof(float)*local_size*local_size, hipMemcpyHostToDevice));
	hipArray *a_gpu;
	checkCudaErrors(hipMallocArray(&a_gpu, &cf, local_size, local_size));
	checkCudaErrors(hipMemcpyToArray(a_gpu, 0, 0, a, sizeof(float)*local_size*local_size, hipMemcpyHostToDevice));
	float *s_m_gpu;
	size_t pitch_s;
	checkCudaErrors(hipMallocPitch(&s_m_gpu, &pitch_s, local_size*((int)s_m_resolution)*sizeof(float), local_size*((int)s_m_resolution)));
	checkCudaErrors(hipMemcpy2D(s_m_gpu, pitch_s, s_m, local_size*((int)s_m_resolution)*sizeof(float), local_size*((int)s_m_resolution)*sizeof(float), local_size*((int)s_m_resolution), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy2D(s_m, local_size*((int)s_m_resolution)*sizeof(float), s_m_gpu, pitch_s, local_size*((int)s_m_resolution)*sizeof(float), local_size*((int)s_m_resolution), hipMemcpyDeviceToHost));
	FILE *s_m_ff;
	FILE *rad;
	FILE *ang;
	s_m_ff=fopen("sensor.dat", "w");
	rad=fopen("radius.dat", "w");
	ang=fopen("angle.dat", "w");
	if(s_m_ff!=NULL){
		fwrite(s_m, sizeof(float), local_size*((int)s_m_resolution)*local_size*((int)s_m_resolution), s_m_ff);
	}
	if(rad!=NULL)
	{
		fwrite(r, sizeof(float), local_size*local_size, rad);
	}
	if(ang!=NULL)
	{
		fwrite(a, sizeof(float), local_size*local_size, ang);
	}

	fclose(s_m_ff);
	fclose(rad);
	fclose(ang);
	*/
	/*map initialization and texture binding*/
    int width=map_size;
    int height=map_size;
	float res=0.025f;
	float rmax=50.0f;
    float* map;
    size_t pitch;
	checkCudaErrors(hipMallocPitch(&map,&pitch,width*sizeof(float), height));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(mapW), &width, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(mapH), &height, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(resolution), &res, sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(range_max), &rmax, sizeof(float)));
	dim3 numThr(32, 32);
	dim3 numBlocks(width/numThr.x, height/numThr.y);
    initMap <<<numBlocks, numThr>>> (map, width, height, pitch/sizeof(float), 1, 1);
	thrust::device_vector<float> delta_t_v(NUM_PARTICLES);
	thrust::device_vector<float> delta_r1_v(NUM_PARTICLES);
	thrust::device_vector<float> delta_r2_v(NUM_PARTICLES);
	thrust::device_vector<float> temp(NUM_PARTICLES);
	thrust::device_vector<float> x_part(NUM_PARTICLES);
	thrust::device_vector<float> y_part(NUM_PARTICLES);
	thrust::device_vector<float> theta_part(NUM_PARTICLES);
	float * scanScores;
	checkCudaErrors(hipMalloc(&scanScores, NUM_PARTICLES*sizeof(float)));
	thrust::device_ptr<float> weights(scanScores);
	thrust::device_vector<float> resampling_vector(NUM_PARTICLES);
	thrust::device_vector<int> resampled_indices(NUM_PARTICLES);
	hipError_t err=hipGetLastError();
	if (err != hipSuccess) 
		printf("Error: %s\n", hipGetErrorString(err));
	checkCudaErrors(hipDeviceSynchronize());
	float *mapsave;
	hipError_t status=hipHostMalloc(&mapsave, width*height*sizeof(float));
	if(status!=hipSuccess)
		printf("error allocating pinned memory\n");
	size_t pitchSave=sizeof(float)*width;
	checkCudaErrors(hipMemcpy2D(mapsave, pitchSave, map, pitch, width*sizeof(float), height, hipMemcpyDeviceToHost));
	FILE *img;
	img=fopen("mapinit.dat", "w");
	if(img!=NULL){
		fwrite(mapsave, sizeof(float), width*height, img);
		/*int ptrIndex=0;
		for(ptrIndex=0; ptrIndex<width*height; ptrIndex++){
			float elem=mapsave[ptrIndex];
			fprintf(img, "%f", elem);
			if(ptrIndex%width==0 && ptrIndex!=0)
				fprintf(img, "\n");
			else
				fprintf(img, " ");
		}*/
	}
	hipHostFree(mapsave);
	fclose(img);

	/*checkCudaErrors(hipBindTexture2D(0,sensor_model, s_m_gpu, local_size*((int)s_m_resolution), local_size*((int)s_m_resolution), pitch_s));
	checkCudaErrors(hipBindTextureToArray(radius, r_gpu));
	checkCudaErrors(hipBindTextureToArray(angle, a_gpu));
	*/
	/*loading the range readings from file*/
	/*FILE *f;
	f=fopen("fr079.log", "r");*/
	float ares=2*M_PI/360.0f;
	int numReadings=(int)(M_PI*2/ares);
	//float amin=-M_PI;
	float amin=0;
	float areadmin=0.0f;
	int astart=(int)((areadmin-amin)/ares);
	std::vector<int> numScans;
	std::vector<std::vector<float>> scans;
	std::vector<float> xs;
	std::vector<float> ys;
	std::vector<float> thetas;
	bool open=loadLog("fr079.log", numScans, scans, xs, ys, thetas);
	/*float *xs=(float*)malloc(sizeof(float));
	float *ys=(float*)malloc(sizeof(float));
	float *thetas=(float*)malloc(sizeof(float));
	int *numScans=(int*)malloc(sizeof(float));
	float **scans=(float**)malloc(sizeof(float*));
	*/
	//int iter=0;
	//int len=0;
	//if (f!=NULL){
	//	char *buffer=(char*)malloc(4096*sizeof(char));
	//	int line=0;
	//	while(fgets(buffer, 4096, f)){
	//		line++;
	//		int numElem=-1;
	//		sscanf(buffer, "FLASER %d", &numElem);
	//		if (numElem==-1){
	//			continue;
	//		}
	//		numElem+=11;
	//		char **a;
	//		char **res;
	//		res=new char* [numElem];
	//		for(a=res; (*a=mystrsep(&buffer, " "))!=NULL;){
	//			if(**a!='\0')
	//				if(++a>=&res[numElem])
	//					break;
	//		}
	//		int i, j;
	//		numScans[iter]=atoi(res[1]);
	//		float *readings_f=(float*)malloc(numReadings*sizeof(float));
	//		/*for(j=0; j<astart; j++){
	//			readings_f[j]=-1.0;
	//		}*/
	//		for(i=2; i<2+atoi(res[1]); i++){
	//			sscanf(res[i], "%f", &readings_f[i-2]);
	//			//readings_f[astart+i-2]*=100;
	//		}
	//		float x=(float)atof(res[i]);
	//		//float x=(float)atof(res[i])*10;
	//		xs[iter]=x;
	//		//float y=(float)atof(res[i+1])*10;
	//		float y=(float)atof(res[i+1]);
	//		ys[iter]=y;
	//		float theta=(float)atof(res[i+2]);
	//		thetas[iter]=theta;
	//		scans[iter]=readings_f;
	//		iter++;
	//		float *xs_new=(float*)realloc(xs, (iter+1)*sizeof(float));
	//		float *ys_new=(float*)realloc(ys, (iter+1)*sizeof(float));
	//		float *thetas_new=(float*)realloc(thetas, (iter+1)*sizeof(float));
	//		int *numScans_new=(int*)realloc(numScans, (iter+1)*sizeof(int));
	//		float **scans_new=(float**)realloc(scans, (iter+1)*sizeof(float*));
	//		if (xs_new!=NULL)
	//			xs=xs_new;
	//		else
	//			printf("no xs");
	//		if (ys_new!=NULL)
	//			ys=ys_new;
	//		else
	//			printf("no ys");
	//		if (thetas_new!=NULL)
	//			thetas=thetas_new;
	//		else
	//			printf("no thetas");
	//		if (scans_new!=NULL)
	//			scans=scans_new;
	//		if(numScans_new!=NULL)
	//			numScans=numScans_new;
	//		else
	//			printf("no scans");
	//		
	//		buffer=(char*)malloc(4096*sizeof(char));
	//	}
	//	xs=(float*)realloc(xs, iter*sizeof(float));
	//	ys=(float*)realloc(ys, iter*sizeof(float));
	//	thetas=(float*)realloc(thetas, iter*sizeof(float));
	//	numScans=(int*)realloc(numScans, iter*sizeof(int));
	//	scans=(float**)realloc(scans, iter*sizeof(float*));
	//	len=iter;
	//	/*int j;
	//	for(j=0; j<iter; j++){
	//		printf("xs:%f\t", xs[j]);
	//		printf("ys:%f\t", ys[j]);
	//		printf("thetas:%f\n", thetas[j]);
	//		int k;
	//		for(k=0; k<numReadings; k++){
	//			float * s=scans[j];
	//			printf("%f\t", s[k]);
	//		}
	//		printf("\n");
	//	}
	//	printf("lines read:%d\n", line);
	//	*/
	//}
	int index;
	float tot_time=0.0f;
	float x_old=0.0f;
	float y_old=0.0f;
	float theta_old=0.0f;
	float x_old_c=0.0f;
	float y_old_c=0.0f;
	float theta_old_c=0.0f;
	printf("%d, %d\n",numScans.size(), open==0);
    for(index=0; index<numScans.size(); index++){
		/*taking one range reading at a time*/
		hipEvent_t start, stop, resample_time;
		hipEvent_t startScores, stopScores;
		float time;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventCreate(&resample_time);
		hipEventCreate(&startScores);
		hipEventCreate(&stopScores);
		hipEventRecord(start, 0);
        std::vector<float> scan=scans[index];
		float x_h;
		float y_h;
		float theta_h;
		/*checkCudaErrors(hipHostMalloc(&x_h, sizeof(float)));
		checkCudaErrors(hipHostMalloc(&y_h, sizeof(float)));
		checkCudaErrors(hipHostMalloc(&theta_h, sizeof(float)));*/
        x_h=xs[index];
        y_h=ys[index];
        theta_h=thetas[index];
		printf("position:%f %f %f\n", x_h, y_h, theta_h);
		float *scan_gpu;
		checkCudaErrors(hipMalloc(&scan_gpu, sizeof(float)*numReadings));
		checkCudaErrors(hipMemcpy(scan_gpu, &scan[0], numReadings*sizeof(float), hipMemcpyHostToDevice));
		/*int numTU=32;
		int numBU=(int)ceil((float)local_size/numTU);
		printf("num blocks:%d\n", numBU);
		dim3 numThrU(numTU, numTU);
		dim3 numBlU(numBU, numBU);
		*/
		/*checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(x), &x_h, sizeof(float)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(y), &y_h, sizeof(float)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(theta), &theta_h, sizeof(float)));
		*/
		//updateMap<<<numBlU, numThrU>>>(x, y, theta*M_PI/180.0f, map, scan_gpu, pitch/sizeof(float), width, height, local_size);
		float delta_t=hypot(x_h-x_old, y_h-y_old);
		float delta_r1=atan2(y_h-y_old, x_h-x_old);
		float delta_r2=theta_h-theta_old-delta_r1;
		float sigma_t=ALPHA3*delta_t+ALPHA4*(fabs(delta_r1)+fabs(delta_r2));
		float sigma_r1=ALPHA1*fabs(delta_r1)+ALPHA2*delta_t;
		float sigma_r2=ALPHA1*fabs(delta_r2)+ALPHA2*delta_t;
		thrust::counting_iterator<unsigned int> rndSeed((int)start);
		thrust::transform(rndSeed, rndSeed+NUM_PARTICLES, delta_t_v.begin(), pseudorgnorm(delta_t, sigma_t));
		thrust::transform(rndSeed+NUM_PARTICLES, rndSeed+NUM_PARTICLES*2, delta_r1_v.begin(), pseudorgnorm(delta_r1, sigma_r1));
		thrust::transform(rndSeed+NUM_PARTICLES*2, rndSeed+NUM_PARTICLES*3, delta_r2_v.begin(), pseudorgnorm(delta_r2, sigma_r2));
		thrust::transform(delta_r1_v.begin(), delta_r1_v.end(), delta_r2_v.begin(), theta_part.begin(), thrust::plus<float>());
		thrust::constant_iterator<float> theta_const(theta_old_c);
		thrust::transform(theta_const, theta_const+NUM_PARTICLES, theta_part.begin(), theta_part.begin(), thrust::plus<float>());
		//thrust::transform(delta_r1_v.begin(), delta_r1_v.end(), x_part.begin(), cos_v());
		thrust::transform(delta_t_v.begin(), delta_t_v.end(), make_transform_iterator(delta_r1_v.begin(), cos_v<float>()), x_part.begin(), thrust::multiplies<float>());
		thrust::constant_iterator<float> x_const(x_old_c);
		thrust::transform(x_const, x_const+NUM_PARTICLES, x_part.begin(), x_part.begin(), thrust::plus<float>());
		//thrust::transform(delta_r1_v.begin(), delta_r1_v.end(), y_part.begin(), sin_v());
		thrust::transform(delta_t_v.begin(), delta_t_v.end(), make_transform_iterator(delta_r1_v.begin(), sin_v<float>()), y_part.begin(), thrust::multiplies<float>());
		thrust::constant_iterator<float> y_const(y_old_c);
		thrust::transform(y_const, y_const+NUM_PARTICLES, y_part.begin(), y_part.begin(), thrust::plus<float>());
		
		float * x_part_kernel=thrust::raw_pointer_cast(&x_part[0]);
		float * y_part_kernel=thrust::raw_pointer_cast(&y_part[0]);
		float * theta_part_kernel=thrust::raw_pointer_cast(&theta_part[0]);
		
		dim3 blocksScores(numScans[index], 256);
		//computeMatchScores<<<blocksScores, NUM_PARTICLES/PART_PER_THREAD >>>(x_part_kernel, y_part_kernel, theta_part_kernel, scan_gpu, map, pitch/sizeof(float), scanScores);
		computeMatchScores<<<NUM_PARTICLES, numScans[index] >>>(x_part_kernel, y_part_kernel, theta_part_kernel, scan_gpu, map, pitch/sizeof(float), scanScores);
		/*thrust::device_vector<float> weights(NUM_PARTICLES);
		thrust::device_vector<float> scoreIndexes(NUM_PARTICLES);
		thrust::reduce_by_key(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), lin_to_row_index<int>(numScans[index])), thrust::make_transform_iterator(thrust::counting_iterator<int>(0)+numScans[index]*NUM_PARTICLES, lin_to_row_index<int>(numScans[index])), allScores, scoreIndexes.begin(), weights.begin(), thrust::equal_to<int>(), thrust::plus<float>());
		checkCudaErrors(hipFree(scanScores));
		*/
		float max_w=thrust::reduce(weights, weights+NUM_PARTICLES, -1.0f, thrust::maximum<float>());
		thrust::constant_iterator<float> max_w_const(max_w);
		thrust::transform(weights, weights+NUM_PARTICLES, max_w_const, weights, thrust::divides<float>());
		/*thrust::constant_iterator<float> one_const(1.0f);
		thrust::transform(one_const, one_const+NUM_PARTICLES, weights.begin(), weights.begin(), thrust::minus<float>());
		*/
		zipIteratorFloatTuple zipIter=thrust::make_zip_iterator(make_tuple(x_part.begin(), y_part.begin(), theta_part.begin()));
		thrust::sort_by_key(weights, weights+NUM_PARTICLES, zipIter);
		thrust::inclusive_scan(weights, weights+NUM_PARTICLES, weights);
		hipEventRecord(resample_time, 0);
		thrust::counting_iterator<unsigned int> resampleSeed((unsigned int)resample_time);
		
		thrust::transform(resampleSeed, resampleSeed+NUM_PARTICLES, resampling_vector.begin(), pseudorg(0.0f, 1.0f));
		thrust::lower_bound(weights, weights+NUM_PARTICLES, resampling_vector.begin(), resampling_vector.end(), resampled_indices.begin());
		thrust::gather(resampled_indices.begin(), resampled_indices.end(), zipIter, zipIter);
		float x_avg=thrust::reduce(x_part.begin(), x_part.end());
		float y_avg=thrust::reduce(y_part.begin(), y_part.end());
		float theta_avg=thrust::reduce(theta_part.begin(), theta_part.end());
		x_avg/=NUM_PARTICLES;
		y_avg/=NUM_PARTICLES;
		theta_avg/=NUM_PARTICLES;
		printf("computed position: %f %f %f\n", x_avg, y_avg, theta_avg);
		updateMapBresenham<<<360, 256>>>(map, pitch/sizeof(float),scan_gpu, x_avg, y_avg, theta_avg);
		checkCudaErrors(hipFree(scan_gpu));
		//checkCudaErrors(hipFree(scanScores));
		/*checkCudaErrors(hipHostFree(x_h));
		checkCudaErrors(hipHostFree(y_h));
		checkCudaErrors(hipHostFree(theta_h));*/
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);
		tot_time+=time;
		hipError_t err=hipGetLastError();
		if (err != hipSuccess){ 
			printf("Error: %s\n", hipGetErrorString(err));
			return -1;
		}
		x_old=x_h;
		y_old=y_h;
		theta_old=theta_h;
		x_old_c=x_avg;
		y_old_c=y_avg;
		theta_old_c=theta_avg;
		if(index%100==0){
			float *mapsave;
			/*saving map at every iteration, just for testing purposes*/
			hipHostMalloc(&mapsave, width*height*sizeof(float));
			size_t pitchSave=sizeof(float)*width;
			checkCudaErrors(hipMemcpy2D(mapsave, pitchSave, map, pitch, width*sizeof(float), height, hipMemcpyDeviceToHost));
			FILE *img;
			char filename[40];
			sprintf(filename, "map%d.dat", index);
			img=fopen(filename, "wb");
			if(img!=NULL){
				fwrite(mapsave, sizeof(float), width*height, img);
				/*int ptrIndex=0;
				for(ptrIndex=0; ptrIndex<width*height; ptrIndex++){
					float elem=mapsave[ptrIndex];
					fprintf(img, "%f ", elem);
					if(ptrIndex%width==0 && ptrIndex!=0)
						fprintf(img, "\n");
					else
						fprintf(img, " ");
				}*/
			}
			hipHostFree(mapsave);
			fclose(img);
		}
    }
	/*unbinding textures and cleanup*/
	/*checkCudaErrors(hipUnbindTexture(radius));
	checkCudaErrors(hipUnbindTexture(angle));
	checkCudaErrors(hipUnbindTexture(sensor_model));
	checkCudaErrors(hipFreeArray(r_gpu));
	checkCudaErrors(hipFreeArray(a_gpu));
	checkCudaErrors(hipFree(s_m_gpu));
	free(r);
	free(a);
	free(s_m);
	*/
	float avg_time=tot_time/numScans.size();
	printf("avg time:%f\n", avg_time);
	getchar();
}
